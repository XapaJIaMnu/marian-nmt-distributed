#include "hip/hip_runtime.h"
#include "optimizers.h"

#include "kernels/tensor_operators.h"
#include "kernels/thrust_functions.h"

namespace marian {
void Sgd::updateImpl(Tensor params, Tensor grads) {
  Element(_1 -= (multiply_factor*eta_) * _2, params, grads);

  hipStreamSynchronize(0);
}

void Adagrad::updateImpl(Tensor params, Tensor grads) {
  if(!alloc_)
    alloc_ = New<TensorAllocator>(params->getDevice());

  if(!gt_) {
    int elements = params->size();
    alloc_->reserveExact(params->memory()->size());
    alloc_->allocate(gt_, {1, elements});
    gt_->set(0);
  }

  Element(_1 += (_2 * _2), gt_, grads);

  Element(_1 -= ((multiply_factor*eta_) / (Sqrt(_2) + eps_)) * _3, params, gt_, grads);

  hipStreamSynchronize(0);
}

void Adam::updateImpl(Tensor params, Tensor grads) {
  if(!mtAlloc_)
    mtAlloc_ = New<TensorAllocator>(params->getDevice());
  if(!vtAlloc_)
    vtAlloc_ = New<TensorAllocator>(params->getDevice());

  if(!mt_) {
    int elements = params->size();
    mtAlloc_->reserveExact(params->memory()->size());
    mtAlloc_->allocate(mt_, {1, elements});
    mt_->set(0);

    vtAlloc_->reserveExact(params->memory()->size());
    vtAlloc_->allocate(vt_, {1, elements});
    vt_->set(0);
  }

  t_++;
  float denom1 = 1 - std::pow(beta1_, t_);
  float denom2 = 1 - std::pow(beta2_, t_);

  Element(_1 = (beta1_ * _1) + ((1 - beta1_) * _2), mt_, grads);
  Element(_1 = (beta2_ * _1) + ((1 - beta2_) * (_2 * _2)), vt_, grads);

  Element(_1 -= (multiply_factor*eta_) * (_2 / denom1) / (Sqrt(_3 / denom2) + eps_),
          params,
          mt_,
          vt_);

 hipStreamSynchronize(0);
}

void Adam::updateState(Ptr<OptimizerBase> localOpt, size_t shardSize_, int my_id) {
  if(!mtAlloc_ || !vtAlloc_) {
    return; //We don't update the optimizer unless it's initialized
  }
  Tensor remoteMT = localOpt->getMT_();
  Tensor remoteVT = localOpt->getVT_();
  int pos = shardSize_*my_id;
  mt_->copyFrom(remoteMT->subtensor(pos, mt_->size()));
  vt_->copyFrom(remoteVT->subtensor(pos, vt_->size()));
}

Ptr<OptimizerBase> Optimizer(Ptr<Config> options) {
  Ptr<ClipperBase> clipper = nullptr;
  float clipNorm = options->get<double>("clip-norm");
  if(clipNorm > 0)
    clipper = Clipper<Norm>(clipNorm);

  float lrate = options->get<double>("learn-rate");

  std::string opt = options->get<std::string>("optimizer");

  float beta1 = options->get<double>("beta1");
  float beta2 = options->get<double>("beta2");
  float eps = options->get<double>("eps");

  Ptr<OptimizerBase> ret;
  if(opt == "sgd") {
    ret = Optimizer<Sgd>(lrate, keywords::clip = clipper);
  } else if(opt == "adagrad") {
    ret = Optimizer<Adagrad>(lrate, keywords::clip = clipper);
  } else if(opt == "adam") {
    ret = Optimizer<Adam>(lrate, keywords::clip = clipper);
  } else {
    UTIL_THROW2("Unknown optimizer: " << opt);
  }

  ret->setB1(beta1);
  ret->setB2(beta2);
  ret->setEPS(eps);
  return ret;
}

Ptr<OptimizerBase> Optimizer(std::string opt, double lrate, double clipNorm) {
  Ptr<ClipperBase> clipper = nullptr;
  if(clipNorm > 0)
    clipper = Clipper<Norm>(clipNorm);

  if(opt == "sgd") {
    return Optimizer<Sgd>(lrate, keywords::clip = clipper);
  } else if(opt == "adagrad") {
    return Optimizer<Adagrad>(lrate, keywords::clip = clipper);
  } else if(opt == "adam") {
    return Optimizer<Adam>(lrate, keywords::clip = clipper);
  } else {
    UTIL_THROW2("Unknown optimizer: " << opt);
  }
}

}
