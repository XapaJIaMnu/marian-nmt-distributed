#include "hip/hip_runtime.h"
#include "training/graph_group_multinode.h"

namespace marian {

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::setScheduler(Ptr<Scheduler<dataset_type>> scheduler) {
  scheduler_ = scheduler;
  // optimizer has to be registered last to see a change of learning rate
  scheduler_->registerTrainingObserver(scheduler_);

  for (auto opt : gpuShardsOpts_) {
    scheduler_->registerTrainingObserver(opt);
  }
}

template <class Builder>
Tensor MultiNodeAsyncGraphGroup<Builder>::newTensor(int size, int device) {
  Tensor t;
  Ptr<TensorAllocator> allocator = New<TensorAllocator>(device);
  allocator->reserveExact(size * sizeof(float));
  allocator->allocate(t, {1, size});
  allocators_.push_back(allocator);
  return t;
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::initFirstRun(Ptr<data::Batch> batch) {
  // Initialize client graphs (incl. params) and builders
  for (size_t i = 0; i < graphs_.size(); ++i) {
    THREAD_GUARD(
        builders_[i]->build(graphs_[i], batch);
        graphs_[i]->forward();
    );
  }
  hipStreamSynchronize(0);
  // Initialize variables for server shard
  initServerShard();
  // Initialize client variables for inter-node communication
  initRemoteCommunicationVars();
  // Initialize sparse server shard variables and launch server thread if sparse communication enabled
  if (dropRate_) {
    initServerShardSparseVars();
    launchSparseServerShardThread();
  } else {
    launchServerShardThread();
  }
  // Launch compute/communicate overlap threads if enabled
  if (commOverlap_) {
    launchCommOverlapThreads();
  }
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::initMPI() {
#if MPI_FOUND
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_comm_world_size_);
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_my_rank_);
#endif
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::initServerShard() {
  // Initialize server shard sizes for all nodes (remote + current)
  size_t totalParamsGradsSize = graphs_[0]->params()->vals()->size();
  size_t nodeShardSize = ceilf(((float) totalParamsGradsSize) / mpi_comm_world_size_);
  size_t remainingTotalSize = totalParamsGradsSize;
  for (int node = 0; node < mpi_comm_world_size_; node++) {
    size_t size = std::min(nodeShardSize, remainingTotalSize);
    nodeShardSizes_.push_back(size);
    remainingTotalSize -= size;
  }

  // Initialize this shard's params and grads
  size_t thisNodeSize = nodeShardSizes_[mpi_my_rank_];
  size_t gpuShardSize = ceilf(((float) thisNodeSize) / devices_.size());
  size_t offset = 0;

  for (int gpu = 0; gpu < devices_.size(); gpu++) {
    size_t size = std::min(gpuShardSize, thisNodeSize - offset);
    Tensor gpuParams = newTensor(size, devices_[gpu]);
    gpuParams->copyFrom(graphs_[0]->params()->vals()->subtensor(offset, size));
    gpuShardsParams_.push_back(gpuParams);
    gpuShardsGrads_.push_back(newTensor(size, devices_[gpu]));
    gpuShardSizes_.push_back(size);
    offset += size;
  }

  // Initialize full send/receive buffer (if no sparse communication)
  if (!dropRate_) {
    serverShardBuffer_ = std::vector<float>(nodeShardSizes_[mpi_my_rank_]);
  }
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::initServerShardSparseVars() {
  // Initialize sizes of clients of every node in cluster
  setupClientSizesOfNodes();

  // Initialize last communicated parameters and delta buffers for all clients of this shard

  size_t thisNodeSize = nodeShardSizes_[mpi_my_rank_];
  size_t gpuShardSize = ceilf(((float) thisNodeSize) / devices_.size());
  size_t offset = 0;

  for (int gpu = 0; gpu < devices_.size(); gpu++) {
    size_t size = std::min(gpuShardSize, thisNodeSize - offset);

    tmpDeltas_.push_back(newTensor(size, devices_[gpu]));
    int sparseCap = graphs_[0]->params()->vals()->size() * 1.2 * (1.0 - dropRate_); // (Estimated) Max size of sparse buffers

    // Server side
    shardSparseGrads_.push_back(
        SparseTensor(new SparseTensorBase(sparseCap, devices_[gpu]))); // @TODO: Sparse sizes can be optimised further
    tmpSparseDeltas_.push_back(SparseTensor(new SparseTensorBase(sparseCap, devices_[gpu])));
    // Client side
    localSparseGrads_.push_back(SparseTensor(new SparseTensorBase(sparseCap, devices_[gpu])));
    localSparseDeltas_.push_back(SparseTensor(new SparseTensorBase(sparseCap, devices_[gpu])));

    // Initialize parameters communicated with all external clients of this server shard (to compute deltas) + gradient droppers
    std::vector<std::vector<Tensor>> extClientParams; // parameters stored for external clients
    std::vector<std::vector<GradientDrop>> extClientDroppers;
    std::vector<GradientDrop> shardDroppers;
    for (int node = 0; node < mpi_comm_world_size_; node++) {
      std::vector<Tensor> nodeParams;
      std::vector<GradientDrop> nodeDroppers;
      for (int client = 0; client < numberClientsOfNodes_[node]; client++) {
        Tensor clientTensor = newTensor(size, devices_[gpu]);
        clientTensor->copyFrom(graphs_[0]->params()->vals()->subtensor(offset, size)); // Copy initial shard params into tensor
        nodeParams.push_back(clientTensor);
        nodeDroppers.push_back(GradientDrop(new GradientDropBase()));
      }
      extClientParams.push_back(nodeParams);
      extClientDroppers.push_back(nodeDroppers);
      shardDroppers.push_back(GradientDrop(new GradientDropBase()));
    }
    clientsParams_.push_back(extClientParams);
    fetchDroppers_.push_back(extClientDroppers); // fetchDroppers_[shard][node][client]
    gradientDroppers_.push_back(shardDroppers);

    offset += size;
  }

  // Initialize send/receive buffers
  serverShardSparseBuffer1_ = std::vector<int>(nodeShardSizes_[mpi_my_rank_]); // @ TODO: Should actually be sparse(X) instead of X but this causes very sporadic crashes
  serverShardSparseBuffer2_ = std::vector<float>(nodeShardSizes_[mpi_my_rank_]);
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::setupClientsOfNodesAndDevices() {
  int index = 0, node = 0, nClientsSeen = 0;
  numberClientsOfNodes_ = std::vector<int>(mpi_comm_world_size_, 0);
  while (index < multiNodeDevices_.size()) {
    if (numberClientsOfNodes_[node] == 0) {
      numberClientsOfNodes_[node] = multiNodeDevices_[index];
      nClientsSeen = 0;
    } else if (nClientsSeen < numberClientsOfNodes_[node]) {
      if (node == mpi_my_rank_) {
        devices_.push_back(multiNodeDevices_[index]);
      }
      nClientsSeen++;
    } else {
      node++;
      index--;
    }
    index++;
  }
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::setupClientSizesOfNodes() {
  for (int node = 0; node < mpi_comm_world_size_; node++) {
    std::string s = "Node ";
    s += std::to_string(node) + " parameter sharding: ";

    clientSizesOfNodes_.push_back(std::vector<size_t>());
    size_t clientSize = ceilf(((float) nodeShardSizes_[node]) / numberClientsOfNodes_[node]);
    size_t offset = 0;
    for (int client = 0; client < numberClientsOfNodes_[node]; client++) {
      size_t size = min(clientSize, nodeShardSizes_[node] - offset);
      clientSizesOfNodes_[node].push_back(size);
      offset += size;

      s += "shard" + std::to_string(client);
      s += " " + std::to_string(size);
      s += client == numberClientsOfNodes_[node] - 1 ? "" : ", ";
    }
    if (mpi_my_rank_ == 0) { LOG(info)->info(s); } // If node 0, print parameter sharding layout
  }
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::initRemoteCommunicationVars() { // @TODO: Integrate with clients / drop-rate / comm-overlap
  for (int gpu = 0; gpu < devices_.size(); gpu++) {
    size_t size = dropRate_ ? (nodeShardSizes_[mpi_my_rank_] * 3 * (1.0 - min(0.99, dropRate_))) : nodeShardSizes_[mpi_my_rank_];
    if (dropRate_) {
      clientShardSparseBuffer1_.push_back(std::vector<int>(size));
      clientShardSparseBuffer2_.push_back(std::vector<float>(size));
    } else {
      clientCommBufferParams_.push_back(std::vector<float>(size));
      clientCommBufferGrads_.push_back(std::vector<float>(size));
    }
    if (commOverlap_) {
      size_t fullSize = graphs_[0]->params()->vals()->size();
      // Running sum of gradients
      Tensor sumGrads = newTensor(fullSize, devices_[gpu]);
      Element(_1 = 0, sumGrads);
      hipStreamSynchronize(0);
      gpuSummedGrads_.push_back(sumGrads);
      // Communication gradients buffer
      commBufferGrads_.push_back(newTensor(fullSize, devices_[gpu]));
      // Communication parameters buffer
      Tensor bufferParams = newTensor(fullSize, devices_[gpu]);
      bufferParams->copyFrom(graphs_[0]->params()->vals());
      commBufferParams_.push_back(bufferParams);
    }
  }
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::launchServerShardThread() {
#if MPI_FOUND
  serverShardThread_ = new std::thread([this] {
    int nCommunicatingNodes = mpi_comm_world_size_; // keep track of number of nodes still communicating with this shard
    MPI_Status status;
    do {
      // Receive grads from any client
      unsigned long messageInfo[4];
      MPI_Recv(&messageInfo, 4, MPI_UNSIGNED_LONG, MPI_ANY_SOURCE, MPI_TAG_GRAD_PUSH_, MPI_COMM_WORLD, &status);
      if (messageInfo[MSG_INFO_STATUS_] == STATUS_NODE_FINISHED_) {
        nCommunicatingNodes--;
        continue;
      } // register finished node and skip to next loop iteration
      MPI_Recv(serverShardBuffer_.data(), nodeShardSizes_[mpi_my_rank_], MPI_FLOAT, status.MPI_SOURCE, MPI_TAG_GRAD_PUSH_, MPI_COMM_WORLD, MPI_STATUS_IGNORE);

      // Update shard params asynchronously over GPUs
      std::vector<std::thread> threads;
      size_t offset = 0;
      for (int gpu = 0; gpu < devices_.size(); gpu++) {
        size_t size = gpuShardSizes_[gpu];

        threads.emplace_back(std::thread([=](int gpu, size_t offset, size_t size, size_t batchWords) {
          std::lock_guard<std::mutex> guard(mutexGpuShards_[gpu]);

          // Copy grads to appropriate GPU
          hipMemcpy(gpuShardsGrads_[gpu]->data(), &serverShardBuffer_.at(offset), size * sizeof(float), hipMemcpyHostToDevice);
          hipStreamSynchronize(0);

          // Run optimizer on GPU
          if (scale_lr && batchWords > 0) {
            gpuShardsOpts_[gpu]->update(gpuShardsParams_[gpu], gpuShardsGrads_[gpu], batchWords / average_batch_words);
          } else {
            gpuShardsOpts_[gpu]->update(gpuShardsParams_[gpu], gpuShardsGrads_[gpu]);
          }
          hipStreamSynchronize(0);
          // Copy params from GPU
          hipMemcpy(&serverShardBuffer_.at(offset), gpuShardsParams_[gpu]->data(), size * sizeof(float), hipMemcpyDeviceToHost);
          hipStreamSynchronize(0);
        }, gpu, offset, size, messageInfo[MSG_INFO_BATCHWORDS_]));

        offset += size;
      }
      for (auto &&t : threads) { t.join(); }

      // Send updated params to same client
      MPI_Ssend(serverShardBuffer_.data(), nodeShardSizes_[mpi_my_rank_], MPI_FLOAT, status.MPI_SOURCE,
                MPI_TAG_PARAM_PUSH_, MPI_COMM_WORLD);

    } while (nCommunicatingNodes != 0);
  });
#endif
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::synchronizeWithServerShards(Tensor newGrads, Tensor oldParams, int gpu, size_t batchWords, std::mutex *optionalBlockMutex) {
  #if MPI_FOUND
  size_t offset = 0;
  for (int node = 0; node < mpi_comm_world_size_; node++) {
    size_t nodeSize = nodeShardSizes_[node];

    // Update remotely if node != this node
    if (node != mpi_my_rank_) {

      // Copy grads from GPU
      hipMemcpy(clientCommBufferGrads_[gpu].data(), newGrads->subtensor(offset, nodeSize)->data(), nodeSize * sizeof(float), hipMemcpyDeviceToHost);
      hipStreamSynchronize(0);

      {
        std::unique_lock<std::mutex> uniqueAccess = (optionalBlockMutex == nullptr) ? std::unique_lock<std::mutex>() : std::unique_lock<std::mutex>(*optionalBlockMutex, std::try_to_lock); // Lock mutex if provided

        // Send grads to server
        size_t messageInfo[4];
        messageInfo[MSG_INFO_SIZE_] = nodeSize;
        messageInfo[MSG_INFO_CLIENT_] = gpu;
        messageInfo[MSG_INFO_BATCHWORDS_] = batchWords;
        messageInfo[MSG_INFO_STATUS_] = STATUS_NODE_TRAINING_;
        MPI_Ssend(&messageInfo, 4, MPI_UNSIGNED_LONG, node, MPI_TAG_GRAD_PUSH_, MPI_COMM_WORLD);
        MPI_Ssend(clientCommBufferGrads_[gpu].data(), nodeSize, MPI_FLOAT, node, MPI_TAG_GRAD_PUSH_, MPI_COMM_WORLD);

        // Receive updated params from server
        MPI_Recv(clientCommBufferParams_[gpu].data(), nodeSize, MPI_FLOAT, node, MPI_TAG_PARAM_PUSH_, MPI_COMM_WORLD,
                 MPI_STATUS_IGNORE);
      }

      // Copy params to GPU
      hipMemcpy(oldParams->subtensor(offset, nodeSize)->data(), clientCommBufferParams_[gpu].data(), nodeSize * sizeof(float), hipMemcpyHostToDevice);
      hipStreamSynchronize(0);


      // Update locally if node == this node
    } else {
      size_t localOffset = offset;
      std::vector<std::thread> threads;

      for (int gpu = 0; gpu < devices_.size(); gpu++) {
        size_t gpuSize = gpuShardSizes_[gpu];

        threads.emplace_back(std::thread([=](int gpu, size_t offset, size_t size) {
          std::lock_guard<std::mutex> guard(mutexGpuShards_[gpu]);

          // Copy grads to appropriate GPU
          gpuShardsGrads_[gpu]->copyFrom(newGrads->subtensor(offset, size));
          // Run optimizer on GPU
          if (scale_lr && batchWords > 0) {
            gpuShardsOpts_[gpu]->update(gpuShardsParams_[gpu], gpuShardsGrads_[gpu], batchWords / average_batch_words);
          } else {
            gpuShardsOpts_[gpu]->update(gpuShardsParams_[gpu], gpuShardsGrads_[gpu]);
          }
          hipStreamSynchronize(0);
          // Copy params back to current GPU
          oldParams->subtensor(offset, size)->copyFrom(gpuShardsParams_[gpu]);
        }, gpu, localOffset, gpuSize));

        localOffset += gpuSize;
      }
      for (auto &&t : threads) { t.join(); }
    }

    offset += nodeSize;
  }
  #endif
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::launchSparseServerShardThread() {
  #if MPI_FOUND
  serverShardThread_ = new std::thread([this] {
    int nCommunicatingNodes = mpi_comm_world_size_; // keep track of number of nodes still communicating with this shard
    MPI_Status status;
    do {
      // Receive sparse grads from any client
      unsigned long messageInfo[4];
      MPI_Recv(&messageInfo, 4, MPI_UNSIGNED_LONG, MPI_ANY_SOURCE, MPI_TAG_GRAD_PUSH_SPARSE1_, MPI_COMM_WORLD, &status);
      if (messageInfo[MSG_INFO_STATUS_] == STATUS_NODE_FINISHED_) {
        nCommunicatingNodes--;
        continue;
      } // register finished node and skip to next loop iteration
      MPI_Recv(serverShardSparseBuffer1_.data(), serverShardSparseBuffer1_.size(), MPI_INT, status.MPI_SOURCE,
               MPI_TAG_GRAD_PUSH_SPARSE2_, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
      MPI_Recv(serverShardSparseBuffer2_.data(), serverShardSparseBuffer2_.size(), MPI_FLOAT, status.MPI_SOURCE,
               MPI_TAG_GRAD_PUSH_SPARSE3_, MPI_COMM_WORLD, MPI_STATUS_IGNORE);

      std::vector<std::thread> threads;
      size_t offset = 0;
      for (int gpu = 0; gpu < devices_.size(); gpu++) {
        size_t endOffset = offset;
        while (endOffset < messageInfo[MSG_INFO_SIZE_] &&
               serverShardSparseBuffer1_.at(endOffset) < gpu * gpuShardSizes_[0] + gpuShardSizes_[gpu]) {
          endOffset++;
        }

        threads.emplace_back(std::thread([=](int gpu, int offset, int size, int client, int batchWords) {

          // Copy sparse grads to appropriate GPU
          hipMemcpy(shardSparseGrads_[gpu]->indices(), &serverShardSparseBuffer1_.at(offset), size * sizeof(int),
                     hipMemcpyHostToDevice);
          hipMemcpy(shardSparseGrads_[gpu]->data(), &serverShardSparseBuffer2_.at(offset), size * sizeof(float),
                     hipMemcpyHostToDevice);
          shardSparseGrads_[gpu]->setSize(size);
          hipStreamSynchronize(0);

          // Convert back to dense, for all index + offset >= 0
          shardSparseGrads_[gpu]->toDense(gpuShardsGrads_[gpu], -(gpuShardSizes_[0] * gpu));
          hipStreamSynchronize(0);

          // Run optimizer on GPU
          if (scale_lr && batchWords > 0) {
            gpuShardsOpts_[gpu]->update(gpuShardsParams_[gpu], gpuShardsGrads_[gpu], batchWords / average_batch_words);
          } else {
            gpuShardsOpts_[gpu]->update(gpuShardsParams_[gpu], gpuShardsGrads_[gpu]);
          }
          hipStreamSynchronize(0);

          // Get deltas = params latest version - params local version
          Element(_1 = _2 - _3, tmpDeltas_[gpu], gpuShardsParams_[gpu], clientsParams_[gpu][status.MPI_SOURCE][client]);
          hipStreamSynchronize(0);

          // Get sparse deltas
          fetchDroppers_[gpu][status.MPI_SOURCE][client]->dropGraph(tmpDeltas_[gpu], tmpSparseDeltas_[gpu], dropRate_);
          // Update shard's last communicated parameters for node's client
          clientsParams_[gpu][status.MPI_SOURCE][client]->copyFrom(gpuShardsParams_[gpu]);

        }, gpu, offset, endOffset - offset, messageInfo[MSG_INFO_CLIENT_], messageInfo[MSG_INFO_BATCHWORDS_]));

        offset += endOffset;
      }
      for (auto &&t : threads) { t.join(); }

      // Copy sparse deltas from GPU (varying sizes so can't do in previous "thread pool" without losing accuracy)
      threads.clear();
      size_t sparseDeltasOffset = 0;
      for (int gpu = 0; gpu < devices_.size(); gpu++) {

        threads.emplace_back(std::thread([=](int gpu, size_t offset) {
          hipMemcpy(&serverShardSparseBuffer1_.at(offset), tmpSparseDeltas_[gpu]->indices(),
                     tmpSparseDeltas_[gpu]->size() * sizeof(int), hipMemcpyDeviceToHost);
          hipMemcpy(&serverShardSparseBuffer2_.at(offset), tmpSparseDeltas_[gpu]->data(),
                     tmpSparseDeltas_[gpu]->size() * sizeof(float), hipMemcpyDeviceToHost);
          hipStreamSynchronize(0);
        }, gpu, sparseDeltasOffset));

        sparseDeltasOffset += tmpSparseDeltas_[gpu]->size();
      }
      for (auto &&t : threads) { t.join(); }

      // Send sparse deltas back to node
      messageInfo[MSG_INFO_SIZE_] = sparseDeltasOffset;
      MPI_Ssend(&messageInfo, 4, MPI_UNSIGNED_LONG, status.MPI_SOURCE, MPI_TAG_PARAM_PUSH_SPARSE1_, MPI_COMM_WORLD);
      MPI_Ssend(serverShardSparseBuffer1_.data(), messageInfo[MSG_INFO_SIZE_], MPI_INT, status.MPI_SOURCE,
                MPI_TAG_PARAM_PUSH_SPARSE2_, MPI_COMM_WORLD);
      MPI_Ssend(serverShardSparseBuffer2_.data(), messageInfo[MSG_INFO_SIZE_], MPI_FLOAT, status.MPI_SOURCE,
                MPI_TAG_PARAM_PUSH_SPARSE3_, MPI_COMM_WORLD);

    } while (nCommunicatingNodes != 0);
  });
  #endif
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::sparseSynchronizeWithServerShards(Tensor newGrads, Tensor oldParams, int gpu, size_t batchWords, std::mutex *optionalBlockMutex) {
#if MPI_FOUND
  size_t offset = 0;
  for (int node = 0; node < mpi_comm_world_size_; node++) {
    size_t nodeSize = nodeShardSizes_[node];

    // Split sparse grads for node
    Tensor subNewGrads = newGrads->subtensor(offset, nodeSize);
    gradientDroppers_[gpu][node]->dropGraph(subNewGrads, localSparseGrads_[gpu], dropRate_);
    SparseTensor sparseSubNewGrads = localSparseGrads_[gpu];

    // Copy to buffers
    hipMemcpy(clientShardSparseBuffer1_[gpu].data(), sparseSubNewGrads->indices(),
               sparseSubNewGrads->size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(clientShardSparseBuffer2_[gpu].data(), sparseSubNewGrads->data(),
               sparseSubNewGrads->size() * sizeof(float), hipMemcpyDeviceToHost);
    hipStreamSynchronize(0); // @TODO: Use safer memory copy by taking min(sparseSubNewGradsSize, bufferSize)

    unsigned long messageInfo[4];
    {
      std::unique_lock<std::mutex> uniqueAccess = (optionalBlockMutex == nullptr) ? std::unique_lock<std::mutex>()
                                                                                  : std::unique_lock<std::mutex>(
              *optionalBlockMutex, std::try_to_lock); // Lock mutex if provided

      // Send sparse grads to node
      messageInfo[MSG_INFO_SIZE_] = sparseSubNewGrads->size();
      messageInfo[MSG_INFO_CLIENT_] = gpu;
      messageInfo[MSG_INFO_BATCHWORDS_] = batchWords;
      messageInfo[MSG_INFO_STATUS_] = STATUS_NODE_TRAINING_;

      MPI_Ssend(&messageInfo, 4, MPI_UNSIGNED_LONG, node, MPI_TAG_GRAD_PUSH_SPARSE1_, MPI_COMM_WORLD);
      MPI_Ssend(clientShardSparseBuffer1_[gpu].data(), messageInfo[MSG_INFO_SIZE_], MPI_INT, node,
                MPI_TAG_GRAD_PUSH_SPARSE2_, MPI_COMM_WORLD);
      MPI_Ssend(clientShardSparseBuffer2_[gpu].data(), messageInfo[MSG_INFO_SIZE_], MPI_FLOAT, node,
                MPI_TAG_GRAD_PUSH_SPARSE3_, MPI_COMM_WORLD);

      // Receive sparse deltas from node
      MPI_Recv(&messageInfo, 4, MPI_UNSIGNED_LONG, node, MPI_TAG_PARAM_PUSH_SPARSE1_, MPI_COMM_WORLD,
               MPI_STATUS_IGNORE);
      MPI_Recv(clientShardSparseBuffer1_[gpu].data(), clientShardSparseBuffer1_[gpu].size(), MPI_INT, node,
               MPI_TAG_PARAM_PUSH_SPARSE2_, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
      MPI_Recv(clientShardSparseBuffer2_[gpu].data(), clientShardSparseBuffer2_[gpu].size(), MPI_FLOAT, node,
               MPI_TAG_PARAM_PUSH_SPARSE3_, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    }

    // Copy to GPUs
    hipMemcpy(localSparseDeltas_[gpu]->indices(), clientShardSparseBuffer1_[gpu].data(),
               messageInfo[MSG_INFO_SIZE_] * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localSparseDeltas_[gpu]->data(), clientShardSparseBuffer2_[gpu].data(),
               messageInfo[MSG_INFO_SIZE_] * sizeof(float), hipMemcpyHostToDevice);
    hipStreamSynchronize(0);
    localSparseDeltas_[gpu]->setSize(messageInfo[MSG_INFO_SIZE_]);

    // Apply sparse deltas to params
    size_t nodeOffset = 0;
    size_t nodeShardSize = clientSizesOfNodes_[node][0];
    for (int nodeShard = 0; nodeShard < numberClientsOfNodes_[node]; nodeShard++) {
      size_t endOffset = nodeOffset;
      while (endOffset + 1 < messageInfo[MSG_INFO_SIZE_] &&
             clientShardSparseBuffer1_[gpu][endOffset] < clientShardSparseBuffer1_[gpu][endOffset + 1]) {
        endOffset++;
      }
      endOffset++;

      SparseTensorBase(localSparseDeltas_[gpu]->data() + nodeOffset, localSparseDeltas_[gpu]->indices() + nodeOffset,
                       endOffset - nodeOffset, gpu).scatterAdd(oldParams->subtensor(offset, nodeSize),
                                                               nodeShard * nodeShardSize);
      nodeOffset += endOffset;
    }
    hipStreamSynchronize(0);

    offset += nodeSize;
  }
#endif
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::launchCommOverlapThreads() {
#if MPI_FOUND
  for (int gpu = 0; gpu < devices_.size(); gpu++) {
    clientCommThreads_.emplace_back(new std::thread([this](int gpu) {
      do {
        // Wait for GPU (client) to fill buffers pointers
        std::unique_lock<std::mutex> uniqueLock(mutexCommBuffersFilled_[gpu]);
        while (!commBuffersFilled_[gpu]) {
          cvCommBuffersFilled_[gpu].wait(uniqueLock);
        }

        if (stopClientCommThreads_) { break; }

        // Synchronize with server shards
        if (dropRate_) {
          sparseSynchronizeWithServerShards(commBufferGrads_[gpu], commBufferParams_[gpu], gpu,
                                            scale_lr ? gpuCommittedWordCounts_[gpu] : 0,
                                            commOverlapSingleActive_ ? &mutexCommChannel_ : nullptr);
        } else {
          synchronizeWithServerShards(commBufferGrads_[gpu], commBufferParams_[gpu], gpu,
                                      scale_lr ? gpuCommittedWordCounts_[gpu] : 0,
                                      commOverlapSingleActive_ ? &mutexCommChannel_ : nullptr);
        }

        // Indicate that buffers can be read from and filled again
        commBuffersFilled_[gpu] = false;

      } while (!stopClientCommThreads_);
    }, gpu));
  }
#endif
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::execute(Ptr<data::Batch> batch) {
  if (!firstBatchProcessed_) {
    initFirstRun(batch);
    firstBatchProcessed_ = true;
  }

  auto task = [this](Ptr<data::Batch> batch) {
    static size_t i = 0;
    thread_local Ptr<ExpressionGraph> graph;
    thread_local Ptr<Builder> builder;
    thread_local size_t t = 0;
    thread_local size_t numSeenWords = 0;

    thread_local Tensor accGradients;
    thread_local Ptr<TensorAllocator> accAlloc;

    thread_local size_t my_id = 0;

    if (!graph) {
      std::lock_guard<std::mutex> lock(mutexClientInit_);
      my_id = i;
      graph = graphs_[i];
      builder = builders_[i++];
    }

    auto costNode = builder->build(graph, batch);

    graph->forward();
    float cost = costNode->scalar();
    graph->backward();

    // Get batch stats
    size_t batchWords = batch->words();

    Tensor gradients;
    if (!commOverlap_ && tau_ > 1) {
      if (t == 0) {
        accAlloc = New<TensorAllocator>(graph->getDevice());
        accAlloc->reserveExact(graph->params()->grads()->memory()->size());
        accAlloc->allocate(accGradients, graph->params()->grads()->shape());
        accGradients->set(0);
      }

      Element(_1 += _2, accGradients, graph->params()->grads());
      gradients = accGradients;
      numSeenWords += batchWords; // Keep track of how many words we've calculated the error from
    } else {
      gradients = graph->params()->grads();
      numSeenWords = batchWords;
    }

    t++;

    hipStreamSynchronize(0);

    if (!commOverlap_ && t % tau_ == 0) {
      if (dropRate_ && t) {
        sparseSynchronizeWithServerShards(gradients, graph->params()->vals(), my_id, numSeenWords);
      } else {
        synchronizeWithServerShards(gradients, graph->params()->vals(), my_id, numSeenWords);
      }
      numSeenWords = 0;

      if (tau_ > 1) {
        gradients->set(0);
      }
    }

    if (scheduler_) {
      boost::upgrade_lock<boost::shared_mutex> lock(schedulerMutex_);
      {
        boost::upgrade_to_unique_lock<boost::shared_mutex> uniqueLock(lock);
        scheduler_->update(cost, batch);
      }

      if (scheduler_->saving()) {
        boost::upgrade_to_unique_lock<boost::shared_mutex> uniqueLock(lock);
        //if(movingAvg_)
        //  fetchParams(graph->params()->vals(), paramsAvg_);
        this->save(graph);
      }

      if (scheduler_->validating()) {
        boost::upgrade_to_unique_lock<boost::shared_mutex> uniqueLock(lock);
        //if(movingAvg_)
        //  fetchParams(graph->params()->vals(), paramsAvg_);
        scheduler_->validate(graph);
      }
    }

    // Overlapping computations with communication
    if (commOverlap_) {

      // Add computed gradients to local running sum
      Element(_1 = _1 + _2, gpuSummedGrads_[my_id], gradients);
      hipStreamSynchronize(0);
      // Sum up word counts if batch flexible learning rate is enabled
      if (scale_lr) {
        gpuSummedWordCounts_[my_id] += numSeenWords;
      }

      // If reached max number of compute iterations per synchronisation, wait for communication channel to finish syncing
      if (maxNumberComputeIters_ != -1 && ++numberComputeIters_[my_id] >= maxNumberComputeIters_) {
        std::lock_guard<std::mutex> wait(mutexCommBuffersFilled_[my_id]);
        numberComputeIters_[my_id] = 0;
      }

      // If communication channel ready, swap graph's pointers with secondary buffers
      if (!commBuffersFilled_[my_id]) {
        std::unique_lock<std::mutex> tryLock(mutexCommBuffersFilled_[my_id], std::try_to_lock);
        if (tryLock.owns_lock()) {

          // Copy summed grads to communication buffer
          commBufferGrads_[my_id]->copyFrom(gpuSummedGrads_[my_id]);
          // Copy parameters from communication buffer
          graph->params()->vals()->copyFrom(commBufferParams_[my_id]);

          // Commit summed word counts if batch-flexible-lr enabled
          if (scale_lr) {
            gpuCommittedWordCounts_[my_id] = gpuSummedWordCounts_[my_id];
            gpuSummedWordCounts_[my_id] = 0;
          }

          // Notify communication thread that buffers have been read and filled
          commBuffersFilled_[my_id] = true;
          cvCommBuffersFilled_[my_id].notify_one();

          // Apply summed gradients to new parameters
          localOpts_[my_id]->update(graph->params()->vals(), gpuSummedGrads_[my_id]);
          hipStreamSynchronize(0);
          // Clear summed gradients
          Element(_1 = 0, gpuSummedGrads_[my_id]);
          hipStreamSynchronize(0);

          numberComputeIters_[my_id] = 0;
        }

      }

    }

  };

  pool_->enqueue(task, batch);
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::signalFinishedToServerShards() {
#if MPI_FOUND
  unsigned long messageInfo[4];
  messageInfo[MSG_INFO_STATUS_] = STATUS_NODE_FINISHED_;
  for (int node = 0; node < mpi_comm_world_size_; node++) {
    MPI_Ssend(&messageInfo, 4, MPI_UNSIGNED_LONG, node, dropRate_ ? MPI_TAG_GRAD_PUSH_SPARSE1_ : MPI_TAG_GRAD_PUSH_,
              MPI_COMM_WORLD);
  }
#endif
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::shutDownServerShardThread() {
  serverShardThread_->join(); // Wait for server thread to finish communicating (with unfinished nodes)
}

template <class Builder>
void MultiNodeAsyncGraphGroup<Builder>::shutDownCommOverlapThreads() {
  stopClientCommThreads_ = true;
  for (int gpu = 0; gpu < devices_.size(); gpu++) {
    commBuffersFilled_[gpu] = true;
    cvCommBuffersFilled_[gpu].notify_one(); // Unblock thread from lock, then join it
    clientCommThreads_[gpu]->join();
  }
}

}